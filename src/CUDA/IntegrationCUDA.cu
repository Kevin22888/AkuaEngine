#include "hip/hip_runtime.h"
#include <AkuaEngine/CUDA/IntegrationCUDA.h>
#include <AkuaEngine/CUDA/SmoothingKernelsCUDA.h>
#include <AkuaEngine/CUDA/MathUtilsCUDA.h>
#include <AkuaEngine/Simulation/Particle.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <thrust/device_vector.h>
#include <glm/glm.hpp>

namespace {

// TODO: move this to a larger collision detection module
// and replace glm::vec3 with float3.
struct CollisionPlane {
    glm::vec3 point;
    glm::vec3 normal;
};

}

namespace AkuaEngine {

namespace IntegrationCUDA {

// =================================== CUDA ====================================

__global__ void kernel_predict_position(Particle* particles, int numParticles, float dt, glm::vec3 force) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i >= numParticles) return;

    Particle* p = &particles[i];

    // The simplest Euler integration
    p->new_velocity = p->velocity + force * dt;
    p->new_position = p->position + p->new_velocity * dt;
}

__global__ void kernel_update_position_and_velocity(Particle* sortedParticles, int numParticles, float dt) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i >= numParticles) return;

    Particle* p = &sortedParticles[i];

    // Update velocity using the corrected position: v_i = (new_position - position) / dt
    p->new_velocity = (p->new_position - p->position) / dt;

    p->position = p->new_position;
    p->velocity = p->new_velocity;
}

__device__ void resolve_collision(
    glm::vec3& position,
    glm::vec3& velocity,
    const glm::vec3& planePoint,
    const glm::vec3& normal,
    float minDist,
    float restitution,
    float friction
) {
    float distance = glm::dot(position - planePoint, normal);
    float approaching = glm::dot(velocity, normal);

    if (distance < minDist) {
        glm::vec3 v_n = approaching * normal;
        glm::vec3 v_t = velocity - v_n;
        
        if (approaching < 0.0f) {    
            velocity = -restitution * v_n + (1.0f - friction) * v_t;
        } else if (fabs(approaching) < 1e-5f) {
            velocity = (1.0f - friction) * v_t;
        }
    }
}

__global__ void kernel_apply_boundary_velocity_damping(
    Particle* sortedParticles,
    int numParticles,
    glm::vec3 boxMin,
    glm::vec3 boxMax,
    float restitution,
    float friction
) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i >= numParticles) return;

    Particle* p = &sortedParticles[i];

    const float minDist = 0.025f;

    CollisionPlane planes[6] = {
        { glm::vec3(boxMin.x, 0, 0),  glm::vec3( 1,  0,  0) },
        { glm::vec3(boxMax.x, 0, 0),  glm::vec3(-1,  0,  0) },
        { glm::vec3(0, boxMin.y, 0),  glm::vec3( 0,  1,  0) },
        { glm::vec3(0, boxMax.y, 0),  glm::vec3( 0, -1,  0) },
        { glm::vec3(0, 0, boxMin.z),  glm::vec3( 0,  0,  1) },
        { glm::vec3(0, 0, boxMax.z),  glm::vec3( 0,  0, -1) }
    };

    for (const auto& plane : planes) {
        resolve_collision(p->position, p->velocity, plane.point, plane.normal, minDist, restitution, friction);
    }
}

__global__ void kernel_compute_vorticities(
    Particle* sortedParticles, 
    int numParticles,
    uint32_t* neighbourArray, 
    uint32_t* neighbourCount,
    float smoothRadius,
    const int maxNeighbours
) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i >= numParticles) return;

    Particle* p_i = &sortedParticles[i];

    p_i->vorticity = glm::vec3(0.0f);

    int start = i * maxNeighbours;
    for (int offset = 0; offset < neighbourCount[i]; offset++) {
        uint32_t j = neighbourArray[start + offset];
        Particle* p_j = &sortedParticles[j];

        glm::vec3 v_ij = p_j->velocity - p_i->velocity;
        glm::vec3 separation = p_i->new_position - p_j->new_position;
        p_i->vorticity += - p_j->mass * MathUtilsCUDA::cross(v_ij, SmoothingKernels::device_gradient_spiky(separation, smoothRadius));
    }
}

__global__ void kernel_apply_vorticity_confinement(
    Particle* sortedParticles,
    int numParticles,
    uint32_t* neighbourArray,
    uint32_t* neighbourCount,
    int maxNeighbours,
    float smoothRadius,
    float deltaTime,
    float vorticityEpsilon
) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i >= numParticles) return;

    Particle* p_i = &sortedParticles[i];

    float inverseDensity = 1 / p_i->density;
    glm::vec3 eta_i = glm::vec3(0.0f);
    int start = i * maxNeighbours;
    for (int offset = 0; offset < neighbourCount[i]; offset++) {
        uint32_t j = neighbourArray[start + offset];
        Particle* p_j = &sortedParticles[j];

        float vorticityDiff = glm::length(p_i->vorticity) - glm::length(p_j->vorticity);
        glm::vec3 separation = p_i->new_position - p_j->new_position;
        eta_i += p_j->mass * vorticityDiff * SmoothingKernels::device_gradient_spiky(separation, smoothRadius);
    }
    eta_i *= inverseDensity;

    float etaLen = glm::length(eta_i);
    if (etaLen < 1e-5f) return;
    glm::vec3 N = eta_i / etaLen;

    glm::vec3 vorticity_force = vorticityEpsilon * MathUtilsCUDA::cross(N, p_i->vorticity);

    p_i->velocity = p_i->velocity + deltaTime * vorticity_force;
}

__global__ void kernel_apply_xsph_viscosity(
    Particle* sortedParticles,
    int numParticles,
    uint32_t* neighbourArray,
    uint32_t* neighbourCount,
    int maxNeighbours,
    float smoothRadius,
    float xsphCoefficient
) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i >= numParticles) return;

    Particle* p_i = &sortedParticles[i];

    glm::vec3 velocity_delta(0.0f);
    int start = i * maxNeighbours;
    for (int offset = 0; offset < neighbourCount[i]; offset++) {
        uint32_t j = neighbourArray[start + offset];
        Particle* p_j = &sortedParticles[j];

        glm::vec3 v_ij = p_j->velocity - p_i->velocity;
        glm::vec3 separation = p_i->new_position - p_j->new_position;
        float w = SmoothingKernels::device_poly6(glm::dot(separation, separation), smoothRadius);

        velocity_delta += (p_j->mass / p_j->density) * v_ij * w;
    }

    p_i->velocity += xsphCoefficient * velocity_delta;
}

// ================================= Wrappers ==================================

void predictNewPositionCUDA(hipGraphicsResource* particlesResource, int numParticles, glm::vec3 force, float deltaTime) {
    // Map resource data to CUDA, retrieve dvice pointer
    hipGraphicsMapResources(1, &particlesResource, 0);
    Particle* d_particles;
    size_t numBytes;
    hipGraphicsResourceGetMappedPointer((void**)&d_particles, &numBytes, particlesResource);

    // Make initial prediction
    int blockSize = 256;
    int gridSize = (numParticles + blockSize - 1) / blockSize;
    kernel_predict_position<<<gridSize, blockSize>>>(d_particles, numParticles, deltaTime, force);
    hipDeviceSynchronize();

    // Let OpenGL regain control over the resource
    hipGraphicsUnmapResources(1, &particlesResource, 0);
}

void updatePositionAndVelocityCUDA(hipGraphicsResource* particlesResource, int numParticles, float deltaTime) {
    // Map resource data to CUDA, retrieve dvice pointer
    hipGraphicsMapResources(1, &particlesResource, 0);
    Particle* d_particles;
    size_t numBytes;
    hipGraphicsResourceGetMappedPointer((void**)&d_particles, &numBytes, particlesResource);

    // Update final position (after constraint solver)
    int blockSize = 256;
    int gridSize = (numParticles + blockSize - 1) / blockSize;
    kernel_update_position_and_velocity<<<gridSize, blockSize>>>(d_particles, numParticles, deltaTime);
    hipDeviceSynchronize();

    hipGraphicsUnmapResources(1, &particlesResource, 0);
}

void applyBoundaryVelocityDampingCUDA(
    hipGraphicsResource* particlesResource,
    int numParticles,
    glm::vec3 boxMin,
    glm::vec3 boxMax,
    float restitution,
    float friction
) {
    hipGraphicsMapResources(1, &particlesResource, 0);
    Particle* d_particles;
    size_t numBytes;
    hipGraphicsResourceGetMappedPointer((void**)&d_particles, &numBytes, particlesResource);

    int blockSize = 256;
    int gridSize = (numParticles + blockSize - 1) / blockSize;
    kernel_apply_boundary_velocity_damping<<<gridSize, blockSize>>>(d_particles, numParticles, boxMin, boxMax, restitution, friction);
    hipDeviceSynchronize();

    hipGraphicsUnmapResources(1, &particlesResource, 0);
}

void applyVorticityAndViscosityCUDA(
    hipGraphicsResource* particlesResource, 
    int numParticles,
    uint32_t* neighbourArray, 
    uint32_t* neighbourCount,
    float smoothRadius,
    float deltaTime,
    float vorticityEpsilon,
    float viscosity,
    const int maxNeighbours
) {
    hipGraphicsMapResources(1, &particlesResource, 0);
    Particle* d_particles;
    size_t numBytes;
    hipGraphicsResourceGetMappedPointer((void**)&d_particles, &numBytes, particlesResource);

    thrust::device_vector<uint32_t> d_neighbourArrayVector(neighbourArray, neighbourArray + numParticles * maxNeighbours);
    thrust::device_vector<uint32_t> d_neighbourCountVector(neighbourCount, neighbourCount + numParticles);
    uint32_t* d_neighbourArray = thrust::raw_pointer_cast(d_neighbourArrayVector.data());
    uint32_t* d_neighbourCount = thrust::raw_pointer_cast(d_neighbourCountVector.data());

    int blockSize = 256;
    int gridSize = (numParticles + blockSize - 1) / blockSize;
    kernel_compute_vorticities<<<gridSize, blockSize>>>(d_particles, numParticles, d_neighbourArray, d_neighbourCount, smoothRadius, maxNeighbours);
    hipDeviceSynchronize();

    kernel_apply_vorticity_confinement<<<gridSize, blockSize>>>(d_particles, numParticles, d_neighbourArray, d_neighbourCount, maxNeighbours, smoothRadius, deltaTime, vorticityEpsilon);
    hipDeviceSynchronize();

    kernel_apply_xsph_viscosity<<<gridSize, blockSize>>>(d_particles, numParticles, d_neighbourArray, d_neighbourCount, maxNeighbours, smoothRadius, viscosity);
    hipDeviceSynchronize();
    
    hipGraphicsUnmapResources(1, &particlesResource, 0);
}

} // namespace IntegrationCUDA

} // namespace AkuaEngine